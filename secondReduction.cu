#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cxtimers.h"

typedef struct {
    int maxThreadsPerBlock;
    size_t sharedMemPerBlock;
} GpuProperties;

void getGpuProperties(GpuProperties *gpuProps) {
    int device;
    hipDeviceProp_t prop;

    // 獲取當前設備ID
    hipGetDevice(&device);

    // 獲取設備屬性
    hipGetDeviceProperties(&prop, device);

    // 填充結構
    gpuProps->maxThreadsPerBlock = prop.maxThreadsPerBlock;
    gpuProps->sharedMemPerBlock  = prop.sharedMemPerBlock;
}

// 使用泰勒級數展開計算正弦函數

__host__ __device__ inline double sinsum(double x, int terms){

    double term = x;
    double sum  = term;
    double x2 = x*x;

    // 循環計算每一項，並加到總和中
    for(int n = 1; n < terms ; n++){
        term *= -x2 / (double)(2*n*(2*n+1));
        sum += term;
    }
    return sum;
}

// GPU上執行的核心函數，計算一系列步驟中的正弦和
__global__ void gpu_sin(double *sums, int steps, int terms, double step_size) {
    extern __shared__ double sharedSums[];

    int step = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // 確保步驟在範圍內
    if (step < steps) {
        double x = step_size * step;
        sharedSums[tid] = sinsum(x, terms);
    } else {
        sharedSums[tid] = 0.0;
    }
    __syncthreads();

    // 在共享記憶體中進行歸納
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sharedSums[tid] += sharedSums[tid + stride];
        }
        __syncthreads();
    }

    // 將每個 block 的結果寫入全局記憶體
    if (tid == 0) {
        sums[blockIdx.x] = sharedSums[0];
    }
}

// 進行reduction，將陣列上的值總和為一個值
__global__ void reductionKernel(double *data, int n) {
    extern __shared__ double sdata[];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    sdata[tid] = (index < n) ? data[index] : 0;
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    // 把每個block的結果寫入global memory
    if (tid == 0) {
        data[blockIdx.x] = sdata[0];
    }
}


double gpuReduction(double *d_sums, int n, GpuProperties gpuProps) {

    int threads = min(256, gpuProps.maxThreadsPerBlock);
    int blocks = (n + threads - 1) / threads;

    while (blocks > 1) {
        int sharedSize = threads * sizeof(double);
        reductionKernel<<<blocks, threads, sharedSize>>>(d_sums, n);
        hipDeviceSynchronize();

        n = blocks;
        blocks = (n + threads - 1) / threads;
    }

    double result;
    hipMemcpy(&result, d_sums, sizeof(double), hipMemcpyDeviceToHost);
    return (double)result;
}

int main(int argc, char *argv[]) {

    // 獲取GPU屬性
    GpuProperties gpuProps;
    getGpuProperties(&gpuProps);

    int steps = (argc > 1) ? atoi(argv[1]) : 65600;
    int terms = (argc > 2) ? atoi(argv[2]) : 10000;

    double pi = 3.1415952535897323;
    double step_size = pi / (steps - 1);

    double *d_sums;
    double *h_sums = (double *)malloc(sizeof(double) * steps);
    
    cx::timer tim;

    // Allocate memory on the GPU
    hipMalloc(&d_sums, sizeof(double) * steps);
    
    // Execute the kernel
    int threadsForSin  = min(256, gpuProps.maxThreadsPerBlock);  // 確保線程數不超過限制
    int blocksForSin  = (steps + threadsForSin - 1) / threadsForSin;
    
    gpu_sin<<<blocksForSin, threadsForSin, sizeof(double) * threadsForSin>>>(d_sums, steps, terms, step_size);
    
    double gpu_sum = gpuReduction(d_sums, steps, gpuProps);
    double gpu_time = tim.lap_ms();

    // Correction and final calculation
    gpu_sum -= 0.5 * (sinsum(0.0f, terms) + sinsum(pi, terms));
    gpu_sum *= step_size;
    
    printf("gpu sum %.10lf steps %d terms %d time %.3f ms\n",gpu_sum,steps,terms,gpu_time);

    // FREEEEEEEEEEEEEEEEEEEEEEEEEEEEEE
    hipFree(d_sums);
    free(h_sums);

    return 0;
}
